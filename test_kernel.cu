
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void kernel() {
    printf("Hello from GPU!\n");
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
